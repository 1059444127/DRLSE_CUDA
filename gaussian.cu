#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>

#include <stdio.h>

#include <gaussian.cuh>
#include <common.cuh>

__constant__ float d_identity[5*5] =   {0,   0,   0,   0,  0,
                                        0,   0,   0,   0,  0,
                                        0,   0,   1,   0,  0,
                                        0,   0,   0,   0,  0,
                                        0,   0,   0,   0,  0};

__constant__ float d_gaussKernel3[3*3];
__constant__ float d_gaussKernel5[5*5] = {0.003765f, 0.015019f, 0.023792f, 0.015019f, 0.003765f,
                                          0.015019f, 0.059912f, 0.094907f, 0.059912f, 0.015019f,
                                          0.023792f, 0.094907f, 0.150342f, 0.094907f, 0.023792f,
                                          0.015019f, 0.059912f, 0.094907f, 0.059912f, 0.015019f,
                                          0.003765f, 0.015019f, 0.023792f, 0.015019f, 0.003765f};

template<typename T>
__global__ void gaussianKernel(hipSurfaceObject_t input, hipSurfaceObject_t output)
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0;
    T sample;

    for(int i = -2; i <= 2; i++)
    {
        for(int j = -2; j <= 2; j++)
        {
            surf2Dread(&sample, input, (x+i)*sizeof(sample), y+j, hipBoundaryModeClamp);
            sum += sample * d_gaussKernel5[5*(i+2) + (j+2)];
        }
    }

    surf2Dwrite(sum,
                output, x * sizeof(float),
                y,
                hipBoundaryModeClamp);
}

template<typename T, hipChannelFormatKind FK>
__host__ T* applyGaussianFilter(int imageWidth, int imageHeight, T* h_dataDicom)
{
    size_t sizeDicom = imageWidth * imageHeight * sizeof(T);

    // Create a Surface with our image data and copy that data to the device
    hipChannelFormatDesc channelFormatDicom = hipCreateChannelDesc(8 * sizeof(T), 0, 0, 0, FK);
    hipArray* d_arrayDicom;
    eee(hipMallocArray(&d_arrayDicom, &channelFormatDicom, imageWidth, imageHeight));
    eee(hipMemcpyToArray(d_arrayDicom, 0, 0, h_dataDicom, sizeDicom, hipMemcpyHostToDevice));

    hipResourceDesc resDescDicom;
    memset(&resDescDicom, 0, sizeof(resDescDicom));
    resDescDicom.resType = hipResourceTypeArray;
    resDescDicom.res.array.array = d_arrayDicom;

    hipSurfaceObject_t d_surfDicom = 0;
    eee(hipCreateSurfaceObject(&d_surfDicom, &resDescDicom));


    // Create an output surface
    hipArray* d_arrayResult;
    eee(hipMallocArray(&d_arrayResult, &channelFormatDicom, imageWidth, imageHeight));

    hipResourceDesc resDescResult;
    memset(&resDescResult, 0, sizeof(resDescResult));
    resDescResult.resType = hipResourceTypeArray;
    resDescResult.res.array.array = d_arrayResult;

    hipSurfaceObject_t d_surfResult = 0;
    eee(hipCreateSurfaceObject(&d_surfResult, &resDescResult));


    // Run kernel
    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    gaussianKernel<T> <<<grid, block>>>(d_surfDicom, d_surfResult);

    // The synchronize call will force the host to wait for the kernel to finish. If we don't
    // do this, we might get errors on future checks, but that indicate errors in the kernel, which
    // can be confusing
    eee(hipPeekAtLastError());
    eee(hipDeviceSynchronize());

    // Copy results to host
    T* outputHost = (T*)malloc(sizeDicom);
    eee(hipMemcpyFromArray(outputHost, d_arrayResult, 0, 0, sizeDicom, hipMemcpyDeviceToHost));

    // Cleanup
    eee(hipDestroySurfaceObject(d_surfDicom));
    eee(hipDestroySurfaceObject(d_surfResult));
    eee(hipFreeArray(d_arrayDicom));
    eee(hipFreeArray(d_arrayResult));
    eee(hipDeviceReset());

    return outputHost;
}

//Explicit instantiation since the compiler has no idea these will be needed in other compilation units
template __host__ short* applyGaussianFilter<short, hipChannelFormatKindSigned>(int imageWidth, int imageHeight, short* textureData);
template __host__ unsigned short* applyGaussianFilter<unsigned short, hipChannelFormatKindUnsigned>(int imageWidth, int imageHeight, unsigned short* textureData);
