#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>

#include <stdio.h>

#include <gaussian.cuh>
#include <common.cuh>


__global__ void gaussianKernel(hipSurfaceObject_t input, hipSurfaceObject_t output)
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0;
    float sample;

    for(int i = -2; i <= 2; i++)
    {
        for(int j = -2; j <= 2; j++)
        {
            surf2Dread(&sample, input, (x+i)*sizeof(sample), y+j, hipBoundaryModeClamp);
            sum += sample * d_gaussKernel5[5*(i+2) + (j+2)];
        }
    }

    surf2Dwrite(sum,
                output, x * sizeof(float),
                y,
                hipBoundaryModeClamp);
}

__host__ float* applyGaussianFilter(int imageWidth, int imageHeight, float* h_dataDicom)
{
    size_t sizeDicom = imageWidth * imageHeight * sizeof(float);

    // Create a Surface with our image data and copy that data to the device
    hipChannelFormatDesc channelFormatDicom = hipCreateChannelDesc(8 * sizeof(float), 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* d_arrayDicom;
    eee(hipMallocArray(&d_arrayDicom, &channelFormatDicom, imageWidth, imageHeight));
    eee(hipMemcpyToArray(d_arrayDicom, 0, 0, h_dataDicom, sizeDicom, hipMemcpyHostToDevice));

    hipResourceDesc resDescDicom;
    memset(&resDescDicom, 0, sizeof(resDescDicom));
    resDescDicom.resType = hipResourceTypeArray;
    resDescDicom.res.array.array = d_arrayDicom;

    hipSurfaceObject_t d_surfDicom = 0;
    eee(hipCreateSurfaceObject(&d_surfDicom, &resDescDicom));


    // Create an output surface
    hipArray* d_arrayResult;
    eee(hipMallocArray(&d_arrayResult, &channelFormatDicom, imageWidth, imageHeight));

    hipResourceDesc resDescResult;
    memset(&resDescResult, 0, sizeof(resDescResult));
    resDescResult.resType = hipResourceTypeArray;
    resDescResult.res.array.array = d_arrayResult;

    hipSurfaceObject_t d_surfResult = 0;
    eee(hipCreateSurfaceObject(&d_surfResult, &resDescResult));


    // Run kernel
    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    gaussianKernel<<<grid, block>>>(d_surfDicom, d_surfResult);

    // The synchronize call will force the host to wait for the kernel to finish. If we don't
    // do this, we might get errors on future checks, but that indicate errors in the kernel, which
    // can be confusing
    eee(hipPeekAtLastError());
    eee(hipDeviceSynchronize());

    // Copy results to host
    float* outputHost = (float*)malloc(sizeDicom);
    eee(hipMemcpyFromArray(outputHost, d_arrayResult, 0, 0, sizeDicom, hipMemcpyDeviceToHost));

    // Cleanup
    eee(hipDestroySurfaceObject(d_surfDicom));
    eee(hipDestroySurfaceObject(d_surfResult));
    eee(hipFreeArray(d_arrayDicom));
    eee(hipFreeArray(d_arrayResult));

    return outputHost;
}
