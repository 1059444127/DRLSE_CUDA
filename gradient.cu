#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>

#include <stdio.h>

#include <gradient.cuh>
#include <common.cuh>

__constant__ float d_sobelX[5*5] = {1,   2,   0,  -2,  -1,
                                    4,   8,   0,  -8,  -4,
                                    6,  12,   0, -12,  -6,
                                    4,   8,   0,  -8,  -4,
                                    1,   2,   0,  -2,  -1};

__constant__ float d_sobelY[5*5] = {1,   4,   6,   4,   1,
                                    2,   8,  12,   8,   2,
                                    0,   0,   0,   0,   0,
                                   -2,  -8, -12,  -8,  -2,
                                   -1,  -4,  -6,  -4,  -1};

__global__ void sobelKernel(hipSurfaceObject_t input, hipSurfaceObject_t output)
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float sumX = 0;
    float sumY = 0;
    int index = 0;
    float sample;

    for(int i = -2; i <= 2; i++)
    {
        for(int j = -2; j <= 2; j++)
        {
            surf2Dread(&sample, input, (x+i)*sizeof(sample), y+j, hipBoundaryModeClamp);

            index = 5*(i+2) + (j+2);
            sumX += sample * d_sobelX[index];
            sumY += sample * d_sobelY[index];
        }
    }

    surf2Dwrite(sqrt(sumX * sumX + sumY * sumY),
                output, x * sizeof(float),
                y,
                hipBoundaryModeClamp);
}

__host__ float* applySobelFilter(int imageWidth, int imageHeight, float* h_dataDicom)
{
    size_t sizeDicom = imageWidth * imageHeight * sizeof(float);

    // Create a Surface with our image data and copy that data to the device
    hipChannelFormatDesc channelFormatDicom = hipCreateChannelDesc(8 * sizeof(float), 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* d_arrayDicom;
    eee(hipMallocArray(&d_arrayDicom, &channelFormatDicom, imageWidth, imageHeight));
    eee(hipMemcpyToArray(d_arrayDicom, 0, 0, h_dataDicom, sizeDicom, hipMemcpyHostToDevice));

    hipResourceDesc resDescDicom;
    memset(&resDescDicom, 0, sizeof(resDescDicom));
    resDescDicom.resType = hipResourceTypeArray;
    resDescDicom.res.array.array = d_arrayDicom;

    hipSurfaceObject_t d_surfDicom = 0;
    eee(hipCreateSurfaceObject(&d_surfDicom, &resDescDicom));


    // Create an output surface, 32-bit float for x, y and magnitude
    hipChannelFormatDesc channelFormatGrad = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* d_arrayResult;
    eee(hipMallocArray(&d_arrayResult, &channelFormatGrad, imageWidth, imageHeight));

    hipResourceDesc resDescResult;
    memset(&resDescResult, 0, sizeof(resDescResult));
    resDescResult.resType = hipResourceTypeArray;
    resDescResult.res.array.array = d_arrayResult;

    hipSurfaceObject_t d_surfResult = 0;
    eee(hipCreateSurfaceObject(&d_surfResult, &resDescResult));


    // Run kernel
    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    sobelKernel<<<grid, block>>>(d_surfDicom, d_surfResult);

    // The synchronize call will force the host to wait for the kernel to finish. If we don't
    // do this, we might get errors on future checks, but that indicate errors in the kernel, which
    // can be confusing
    eee(hipPeekAtLastError());
    eee(hipDeviceSynchronize());

    // Copy results to host
    float* outputHost = (float*)malloc(sizeDicom); //return 2 channels
    eee(hipMemcpyFromArray(outputHost, d_arrayResult, 0, 0, sizeDicom, hipMemcpyDeviceToHost));

    // Cleanup
    eee(hipDestroySurfaceObject(d_surfDicom));
    eee(hipDestroySurfaceObject(d_surfResult));
    eee(hipFreeArray(d_arrayDicom));
    eee(hipFreeArray(d_arrayResult));
    eee(hipDeviceReset());

    return outputHost;
}
