
#include <hip/hip_runtime.h>

__global__ void simpleKernel(short* output, hipTextureObject_t input, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Convert to texture units
    float u = x / (float)width;
    float v = y / (float)height;

    output[y * width + x] = (short)(0.5 * tex2D<short>(input, u, v));
}

__global__ void simpleKernel(unsigned short* output, hipTextureObject_t input, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Convert to texture units
    float u = x / (float)width;
    float v = y / (float)height;

    output[y * width + x] = (short)(0.5 * tex2D<unsigned short>(input, u, v));
}

__global__ void rasterizerTest(short* output, hipTextureObject_t dicomTex, hipTextureObject_t polylineTex, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Convert to texture units
    float u = x / (float)width;
    float v = y / (float)height;

    unsigned int sum = 0;

    //Set sum to 1 if we're next to a painted square on polylineTex
    short dicomSample = tex2D<short>(dicomTex, u, v);
    for(int i = -1; i < 2; i++)
    {
        for(int j = -1; j < 2; j++)
        {
            sum += tex2D<unsigned char>(polylineTex, (x + i) / (float)width, (y + j) / (float)height);
        }
    }
    sum = min(sum, 1);

    output[y * width + x] = dicomSample * (1 - sum);
}

__global__ void rasterizerTest(unsigned short* output, hipTextureObject_t dicomTex, hipTextureObject_t polylineTex, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Convert to texture units
    float u = x / (float)width;
    float v = y / (float)height;

    unsigned int sum = 0;

    //Set sum to 1 if we're next to a painted square on polylineTex
    unsigned short dicomSample = tex2D<unsigned short>(dicomTex, u, v);
    for(int i = -1; i < 2; i++)
    {
        for(int j = -1; j < 2; j++)
        {
            sum += tex2D<unsigned char>(polylineTex, (x + i) / (float)width, (y + j) / (float)height);
        }
    }
    sum = min(sum, 1);

    output[y * width + x] = dicomSample * (1 - sum);
}

__host__ short* modifyTexture(int imageWidth, int imageHeight, short* textureData)
{
    hipError_t e;
    size_t size = imageWidth * imageHeight * sizeof(short);

    // Allocate a cudaArray in device memory and copy our texture data there
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindSigned);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, imageWidth, imageHeight);
    hipMemcpyToArray(cuArray, 0, 0, textureData, size, hipMemcpyHostToDevice);

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t tex = 0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

    // Create result array in the device
    short* outputDev;
    hipMalloc(&outputDev, size);

    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    simpleKernel<<<grid, block>>>(outputDev, tex, imageWidth, imageHeight);

    // Create result array in the host
    short* outputHost = (short*)malloc(size);

    // Copy results to host
    hipMemcpy(outputHost, outputDev, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroyTextureObject(tex);
    hipFreeArray(cuArray);
    hipFree(outputDev);

    return outputHost;
}

__host__ unsigned short* modifyTexture(int imageWidth, int imageHeight, unsigned short* textureData)
{
    hipError_t e;
    size_t size = imageWidth * imageHeight * sizeof(unsigned short);

    // Allocate a cudaArray in device memory and copy our texture data there
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, imageWidth, imageHeight);
    hipMemcpyToArray(cuArray, 0, 0, textureData, size, hipMemcpyHostToDevice);

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t tex = 0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

    // Create result array in the device
    unsigned short* outputDev;
    hipMalloc(&outputDev, size);

    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    simpleKernel<<<grid, block>>>(outputDev, tex, imageWidth, imageHeight);

    // Create result array in the host
    unsigned short* outputHost = (unsigned short*)malloc(size);

    // Copy results to host
    hipMemcpy(outputHost, outputDev, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroyTextureObject(tex);
    hipFreeArray(cuArray);
    hipFree(outputDev);

    return outputHost;
}

__host__ short* modifyTextureRasterized(int imageWidth, int imageHeight, short* dicomData, unsigned char* polylineData)
{
    hipError_t e;
    size_t sizeDicom = imageWidth * imageHeight * sizeof(short);
    size_t sizePolyline = imageWidth * imageHeight * sizeof(unsigned char);

    // Allocate a cudaArray in device memory and copy our dicomData there
    hipChannelFormatDesc channelDescD = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindSigned);
    hipArray* cuArrayD;
    hipMallocArray(&cuArrayD, &channelDescD, imageWidth, imageHeight);
    hipMemcpyToArray(cuArrayD, 0, 0, dicomData, sizeDicom, hipMemcpyHostToDevice);

    hipResourceDesc resDescD;
    memset(&resDescD, 0, sizeof(resDescD));
    resDescD.resType = hipResourceTypeArray;
    resDescD.res.array.array = cuArrayD;

    hipTextureDesc texDescD;
    memset(&texDescD, 0, sizeof(texDescD));
    texDescD.readMode = hipReadModeElementType;
    texDescD.addressMode[0] = hipAddressModeClamp;
    texDescD.addressMode[1] = hipAddressModeClamp;
    texDescD.filterMode = hipFilterModePoint;
    texDescD.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t texD = 0;
    hipCreateTextureObject(&texD, &resDescD, &texDescD, NULL);

    // Allocate a cudaArray in device memory and copy our polylineData there
    hipChannelFormatDesc channelDescP = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipArray* cuArrayP;
    hipMallocArray(&cuArrayP, &channelDescP, imageWidth, imageHeight);
    hipMemcpyToArray(cuArrayP, 0, 0, polylineData, sizePolyline, hipMemcpyHostToDevice);

    hipResourceDesc resDescP;
    memset(&resDescP, 0, sizeof(resDescP));
    resDescP.resType = hipResourceTypeArray;
    resDescP.res.array.array = cuArrayP;

    hipTextureDesc texDescP;
    memset(&texDescP, 0, sizeof(texDescP));
    texDescP.readMode = hipReadModeElementType;
    texDescP.addressMode[0] = hipAddressModeClamp;
    texDescP.addressMode[1] = hipAddressModeClamp;
    texDescP.filterMode = hipFilterModePoint;
    texDescP.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t texP = 0;
    hipCreateTextureObject(&texP, &resDescP, &texDescP, NULL);

    // Create result array in the device
    short* outputDev;
    hipMalloc(&outputDev, sizeDicom);

    // Create result array in the host
    short* outputHost = (short*)malloc(sizeDicom);

    // Run kernel
    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    rasterizerTest<<<grid, block>>>(outputDev, texD, texP, imageWidth, imageHeight);

    // Copy results to host
    hipMemcpy(outputHost, outputDev, sizeDicom, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroyTextureObject(texD);
    hipDestroyTextureObject(texP);
    hipFreeArray(cuArrayD);
    hipFreeArray(cuArrayP);
    hipFree(outputDev);

    return outputHost;
}

__host__ unsigned short* modifyTextureRasterized(int imageWidth, int imageHeight, unsigned short* dicomData, unsigned char* polylineData)
{
    hipError_t e;
    size_t sizeDicom = imageWidth * imageHeight * sizeof(unsigned short);
    size_t sizePolyline = imageWidth * imageHeight * sizeof(unsigned char);

    // Allocate a cudaArray in device memory and copy our dicomData there
    hipChannelFormatDesc channelDescD = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipArray* cuArrayD;
    hipMallocArray(&cuArrayD, &channelDescD, imageWidth, imageHeight);
    hipMemcpyToArray(cuArrayD, 0, 0, dicomData, sizeDicom, hipMemcpyHostToDevice);

    hipResourceDesc resDescD;
    memset(&resDescD, 0, sizeof(resDescD));
    resDescD.resType = hipResourceTypeArray;
    resDescD.res.array.array = cuArrayD;

    hipTextureDesc texDescD;
    memset(&texDescD, 0, sizeof(texDescD));
    texDescD.readMode = hipReadModeElementType;
    texDescD.addressMode[0] = hipAddressModeClamp;
    texDescD.addressMode[1] = hipAddressModeClamp;
    texDescD.filterMode = hipFilterModePoint;
    texDescD.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t texD = 0;
    hipCreateTextureObject(&texD, &resDescD, &texDescD, NULL);

    // Allocate a cudaArray in device memory and copy our polylineData there
    hipChannelFormatDesc channelDescP = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipArray* cuArrayP;
    hipMallocArray(&cuArrayP, &channelDescP, imageWidth, imageHeight);
    hipMemcpyToArray(cuArrayP, 0, 0, polylineData, sizePolyline, hipMemcpyHostToDevice);

    hipResourceDesc resDescP;
    memset(&resDescP, 0, sizeof(resDescP));
    resDescP.resType = hipResourceTypeArray;
    resDescP.res.array.array = cuArrayP;

    hipTextureDesc texDescP;
    memset(&texDescP, 0, sizeof(texDescP));
    texDescP.readMode = hipReadModeElementType;
    texDescP.addressMode[0] = hipAddressModeClamp;
    texDescP.addressMode[1] = hipAddressModeClamp;
    texDescP.filterMode = hipFilterModePoint;
    texDescP.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t texP = 0;
    hipCreateTextureObject(&texP, &resDescP, &texDescP, NULL);

    // Create result array in the device
    unsigned short* outputDev;
    hipMalloc(&outputDev, sizeDicom);

    // Create result array in the host
    unsigned short* outputHost = (unsigned short*)malloc(sizeDicom);

    // Run kernel
    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    rasterizerTest<<<grid, block>>>(outputDev, texD, texP, imageWidth, imageHeight);

    // Copy results to host
    hipMemcpy(outputHost, outputDev, sizeDicom, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroyTextureObject(texD);
    hipDestroyTextureObject(texP);
    hipFreeArray(cuArrayD);
    hipFreeArray(cuArrayP);
    hipFree(outputDev);

    return outputHost;
}
