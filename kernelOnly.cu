
#include <hip/hip_runtime.h>

__global__ void diagKernel(short* output, hipTextureObject_t input, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Convert to texture units
    float u = x / (float)width;
    float v = y / (float)height;

    output[y * width + x] = (short)(0.5 * tex2D<short>(input, u, v));
}

__global__ void diagKernel(unsigned short* output, hipTextureObject_t input, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Convert to texture units
    float u = x / (float)width;
    float v = y / (float)height;

    output[y * width + x] = (short)(0.5 * tex2D<unsigned short>(input, u, v));
}

__host__ short* modifyTexture(int imageWidth, int imageHeight, short* textureData)
{
    hipError_t e;
    size_t size = imageWidth * imageHeight * sizeof(short);

    // Allocate a cudaArray in device memory and copy our texture data there
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindSigned);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, imageWidth, imageHeight);
    hipMemcpyToArray(cuArray, 0, 0, textureData, size, hipMemcpyHostToDevice);

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t tex = 0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

    // Create result array in the device
    short* outputDev;
    hipMalloc(&outputDev, size);

    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    diagKernel<<<grid, block>>>(outputDev, tex, imageWidth, imageHeight);

    // Create result array in the host
    short* outputHost = (short*)malloc(size);

    // Copy results to host
    hipMemcpy(outputHost, outputDev, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroyTextureObject(tex);
    hipFreeArray(cuArray);
    hipFree(outputDev);

    return outputHost;
}

__host__ unsigned short* modifyTexture(int imageWidth, int imageHeight, unsigned short* textureData)
{
    hipError_t e;
    size_t size = imageWidth * imageHeight * sizeof(unsigned short);

    // Allocate a cudaArray in device memory and copy our texture data there
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, imageWidth, imageHeight);
    hipMemcpyToArray(cuArray, 0, 0, textureData, size, hipMemcpyHostToDevice);

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t tex = 0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

    // Create result array in the device
    unsigned short* outputDev;
    hipMalloc(&outputDev, size);

    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    diagKernel<<<grid, block>>>(outputDev, tex, imageWidth, imageHeight);

    // Create result array in the host
    unsigned short* outputHost = (unsigned short*)malloc(size);

    // Copy results to host
    hipMemcpy(outputHost, outputDev, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroyTextureObject(tex);
    hipFreeArray(cuArray);
    hipFree(outputDev);

    return outputHost;
}
