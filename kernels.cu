#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>

#include <stdio.h>

#include <kernels.cuh>
#include <gaussian.cuh>
#include <common.cuh>

//====================================================================================
//KERNELS
//====================================================================================

__global__ void edgeIndicatorKernel(hipSurfaceObject_t gaussInput, hipSurfaceObject_t output)
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float sumX = 0;
    float sumY = 0;
    int index = 0;
    float sample;

    for(int i = -2; i <= 2; i++)
    {
        for(int j = -2; j <= 2; j++)
        {
            surf2Dread(&sample, gaussInput, (x+i)*sizeof(sample), y+j, hipBoundaryModeClamp);

            index = 5*(i+2) + (j+2);
            sumX += sample * d_sobelX[index];
            sumY += sample * d_sobelY[index];
        }
    }

    surf2Dwrite(1.0f / (1.0f + sumX * sumX + sumY * sumY),
                output, x * sizeof(float),
                y,
                hipBoundaryModeClamp);
}

//====================================================================================
//HOST CUDA FUNCTIONS
//====================================================================================
__host__ float* testEdgeIndicator(int imageWidth, int imageHeight, float* h_dataDicom)
{
    size_t sizeDicom = imageWidth * imageHeight * sizeof(float);

    // Create a Surface with our image data and copy that data to the device
    hipChannelFormatDesc channelFormatDicom = hipCreateChannelDesc(8 * sizeof(float), 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* d_arrayDicom;
    eee(hipMallocArray(&d_arrayDicom, &channelFormatDicom, imageWidth, imageHeight));
    eee(hipMemcpyToArray(d_arrayDicom, 0, 0, h_dataDicom, sizeDicom, hipMemcpyHostToDevice));

    hipResourceDesc resDescDicom;
    memset(&resDescDicom, 0, sizeof(resDescDicom));
    resDescDicom.resType = hipResourceTypeArray;
    resDescDicom.res.array.array = d_arrayDicom;

    hipSurfaceObject_t d_surfDicom = 0;
    eee(hipCreateSurfaceObject(&d_surfDicom, &resDescDicom));


    // Create a temp surface for the gaussian filtered input image
    hipChannelFormatDesc channelFormatGauss = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* d_arrayGauss;
    eee(hipMallocArray(&d_arrayGauss, &channelFormatGauss, imageWidth, imageHeight));

    hipResourceDesc resDescGauss;
    memset(&resDescGauss, 0, sizeof(resDescGauss));
    resDescGauss.resType = hipResourceTypeArray;
    resDescGauss.res.array.array = d_arrayGauss;

    hipSurfaceObject_t d_surfGaussian = 0;
    eee(hipCreateSurfaceObject(&d_surfGaussian, &resDescGauss));


    // Create an output surface for the edge indicator image
    hipChannelFormatDesc channelFormatRes = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* d_arrayResult;
    eee(hipMallocArray(&d_arrayResult, &channelFormatRes, imageWidth, imageHeight));

    hipResourceDesc resDescResult;
    memset(&resDescResult, 0, sizeof(resDescResult));
    resDescResult.resType = hipResourceTypeArray;
    resDescResult.res.array.array = d_arrayResult;

    hipSurfaceObject_t d_surfResult = 0;
    eee(hipCreateSurfaceObject(&d_surfResult, &resDescResult));


    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);

    // Run gaussian kernel
    gaussianKernel<<<grid, block>>>(d_surfDicom, d_surfGaussian);

    // Run kernel
    edgeIndicatorKernel<<<grid, block>>>(d_surfGaussian, d_surfResult);


    // The synchronize call will force the host to wait for the kernel to finish. If we don't
    // do this, we might get errors on future checks, but that indicate errors in the kernel, which
    // can be confusing
    eee(hipPeekAtLastError());
    eee(hipDeviceSynchronize());

    // Copy results to host
    float* outputHost = (float*)malloc(sizeDicom); //return 2 channels
    eee(hipMemcpyFromArray(outputHost, d_arrayResult, 0, 0, sizeDicom, hipMemcpyDeviceToHost));

    // Cleanup
    eee(hipDestroySurfaceObject(d_surfDicom));
    eee(hipDestroySurfaceObject(d_surfGaussian));
    eee(hipDestroySurfaceObject(d_surfResult));
    eee(hipFreeArray(d_arrayDicom));
    eee(hipFreeArray(d_arrayGauss));
    eee(hipFreeArray(d_arrayResult));
    eee(hipDeviceReset());

    return outputHost;
}
