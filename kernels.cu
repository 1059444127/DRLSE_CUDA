#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>

#include <kernels.cuh>

template<typename T>
__global__ void simpleKernel(T* output, hipTextureObject_t input, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Convert to texture units
    float u = x / (float)width;
    float v = y / (float)height;

    output[y * width + x] = (T)(0.5 * tex2D<T>(input, u, v));
}

template<typename T>
__global__ void rasterizerTest(T* output, hipTextureObject_t dicomTex, hipTextureObject_t polylineTex, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Convert to texture units
    float u = x / (float)width;
    float v = y / (float)height;

    unsigned int sum = 0;

    //Set sum to 1 if we're next to a painted square on polylineTex
    T dicomSample = tex2D<T>(dicomTex, u, v);
    for(int i = -1; i < 2; i++)
    {
        for(int j = -1; j < 2; j++)
        {
            sum += tex2D<unsigned char>(polylineTex, (x + i) / (float)width, (y + j) / (float)height);
        }
    }
    sum = min(sum, 1);

    output[y * width + x] = dicomSample * (1 - sum);
}

template<typename T, hipChannelFormatKind FK>
__host__ T* modifyTexture(int imageWidth, int imageHeight, T* textureData)
{
    size_t size = imageWidth * imageHeight * sizeof(T);

    // Allocate a hipArray in device memory and copy our texture data there
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8 * sizeof(T), 0, 0, 0, FK);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, imageWidth, imageHeight);
    hipMemcpyToArray(cuArray, 0, 0, textureData, size, hipMemcpyHostToDevice);

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t tex = 0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

    // Create result array in the device
    T* outputDev;
    hipMalloc(&outputDev, size);

    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    simpleKernel<<<grid, block>>>(outputDev, tex, imageWidth, imageHeight);

    // Create result array in the host
    T* outputHost = (T*)malloc(size);

    // Copy results to host
    hipMemcpy(outputHost, outputDev, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroyTextureObject(tex);
    hipFreeArray(cuArray);
    hipFree(outputDev);

    return outputHost;
}

template<typename T, hipChannelFormatKind FK>
__host__ T* modifyTextureRasterized(int imageWidth, int imageHeight, T* dicomData, unsigned char* polylineData)
{
    size_t sizeDicom = imageWidth * imageHeight * sizeof(T);
    size_t sizePolyline = imageWidth * imageHeight * sizeof(unsigned char);

    // Allocate a hipArray in device memory and copy our dicomData there
    hipChannelFormatDesc channelDescD = hipCreateChannelDesc(8 * sizeof(T), 0, 0, 0, FK);
    hipArray* cuArrayD;
    hipMallocArray(&cuArrayD, &channelDescD, imageWidth, imageHeight);
    hipMemcpyToArray(cuArrayD, 0, 0, dicomData, sizeDicom, hipMemcpyHostToDevice);

    hipResourceDesc resDescD;
    memset(&resDescD, 0, sizeof(resDescD));
    resDescD.resType = hipResourceTypeArray;
    resDescD.res.array.array = cuArrayD;

    hipTextureDesc texDescD;
    memset(&texDescD, 0, sizeof(texDescD));
    texDescD.readMode = hipReadModeElementType;
    texDescD.addressMode[0] = hipAddressModeClamp;
    texDescD.addressMode[1] = hipAddressModeClamp;
    texDescD.filterMode = hipFilterModePoint;
    texDescD.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t texD = 0;
    hipCreateTextureObject(&texD, &resDescD, &texDescD, NULL);

    // Allocate a hipArray in device memory and copy our polylineData there
    hipChannelFormatDesc channelDescP = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipArray* cuArrayP;
    hipMallocArray(&cuArrayP, &channelDescP, imageWidth, imageHeight);
    hipMemcpyToArray(cuArrayP, 0, 0, polylineData, sizePolyline, hipMemcpyHostToDevice);

    hipResourceDesc resDescP;
    memset(&resDescP, 0, sizeof(resDescP));
    resDescP.resType = hipResourceTypeArray;
    resDescP.res.array.array = cuArrayP;

    hipTextureDesc texDescP;
    memset(&texDescP, 0, sizeof(texDescP));
    texDescP.readMode = hipReadModeElementType;
    texDescP.addressMode[0] = hipAddressModeClamp;
    texDescP.addressMode[1] = hipAddressModeClamp;
    texDescP.filterMode = hipFilterModePoint;
    texDescP.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t texP = 0;
    hipCreateTextureObject(&texP, &resDescP, &texDescP, NULL);

    // Create result array in the device
    T* outputDev;
    hipMalloc(&outputDev, sizeDicom);

    // Create result array in the host
    T* outputHost = (T*)malloc(sizeDicom);

    // Run kernel
    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    rasterizerTest<<<grid, block>>>(outputDev, texD, texP, imageWidth, imageHeight);

    // Copy results to host
    hipMemcpy(outputHost, outputDev, sizeDicom, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroyTextureObject(texD);
    hipDestroyTextureObject(texP);
    hipFreeArray(cuArrayD);
    hipFreeArray(cuArrayP);
    hipFree(outputDev);

    return outputHost;
}

//Explicit instantiation so these signatures are available when the linker is linking our lib to the exe
template __host__ short* modifyTexture<short, hipChannelFormatKindSigned>(int imageWidth, int imageHeight, short* textureData);
template __host__ unsigned short* modifyTexture<unsigned short, hipChannelFormatKindUnsigned>(int imageWidth, int imageHeight, unsigned short* textureData);
template __host__ short* modifyTextureRasterized<short, hipChannelFormatKindSigned>(int imageWidth, int imageHeight, short* dicomData, unsigned char* polylineData);
template __host__ unsigned short* modifyTextureRasterized<unsigned short, hipChannelFormatKindUnsigned>(int imageWidth, int imageHeight, unsigned short* dicomData, unsigned char* polylineData);
