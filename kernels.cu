#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>

#include <stdio.h>

#include <kernels.cuh>
#include <common.cuh>

//====================================================================================
//KERNELS
//====================================================================================
template<typename T>
__global__ void simpleKernel(T* output, hipTextureObject_t input, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Convert to texture units
    float u = x / (float)width;
    float v = y / (float)height;

    output[y * width + x] = (T)(0.5 * tex2D<T>(input, u, v));
}

template<typename T>
__global__ void rasterizerTest(T* output, hipTextureObject_t dicomTex, hipTextureObject_t polylineTex, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Convert to texture units
    float u = x / (float)width;
    float v = y / (float)height;

    unsigned int sum = 0;

    //Set sum to 1 if we're next to a painted square on polylineTex
    T dicomSample = tex2D<T>(dicomTex, u, v);
    for(int i = -1; i < 2; i++)
    {
        for(int j = -1; j < 2; j++)
        {
            sum += tex2D<unsigned char>(polylineTex, (x + i) / (float)width, (y + j) / (float)height);
        }
    }
    sum = min(sum, 1);

    output[y * width + x] = dicomSample * (1 - sum);
}

__global__ void rasterizerTestSurface(hipSurfaceObject_t input, hipSurfaceObject_t output, hipSurfaceObject_t polyline, int width, int height)
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 1 && x < width-2 && y > 1 && y < height-2)
    {
        int sum = 0;
        short sample;
        for(int i = -2; i <= 2; i++)
        {
            for(int j = -2; j <= 2; j++)
            {
                surf2Dread(&sample, input, (int)((x+i)*sizeof(sample)), (int)(y+j));
                //sum += sample * d_gaussKernel5[5*i+j];
            }
        }

        surf2Dwrite((short)sum, output, x * sizeof(sample), y);
    }
}

//====================================================================================
//HOST CUDA FUNCTIONS
//====================================================================================
template<typename T, hipChannelFormatKind FK>
__host__ T* modifyTexture(int imageWidth, int imageHeight, T* textureData)
{
    size_t size = imageWidth * imageHeight * sizeof(T);

    // Allocate a hipArray in device memory and copy our texture data there
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8 * sizeof(T), 0, 0, 0, FK);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, imageWidth, imageHeight);
    hipMemcpyToArray(cuArray, 0, 0, textureData, size, hipMemcpyHostToDevice);

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t tex = 0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

    // Create result array in the device
    T* outputDev;
    hipMalloc(&outputDev, size);

    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    simpleKernel<<<grid, block>>>(outputDev, tex, imageWidth, imageHeight);

    // Create result array in the host
    T* outputHost = (T*)malloc(size);

    // Copy results to host
    hipMemcpy(outputHost, outputDev, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroyTextureObject(tex);
    hipFreeArray(cuArray);
    hipFree(outputDev);

    return outputHost;
}

template<typename T, hipChannelFormatKind FK>
__host__ T* modifyTextureRasterized(int imageWidth, int imageHeight, T* dicomData, unsigned char* polylineData)
{
    size_t sizeDicom = imageWidth * imageHeight * sizeof(T);
    size_t sizePolyline = imageWidth * imageHeight * sizeof(unsigned char);

    // Allocate a hipArray in device memory and copy our dicomData there
    hipChannelFormatDesc channelDescD = hipCreateChannelDesc(8 * sizeof(T), 0, 0, 0, FK);
    hipArray* cuArrayD;
    hipMallocArray(&cuArrayD, &channelDescD, imageWidth, imageHeight);
    hipMemcpyToArray(cuArrayD, 0, 0, dicomData, sizeDicom, hipMemcpyHostToDevice);

    hipResourceDesc resDescD;
    memset(&resDescD, 0, sizeof(resDescD));
    resDescD.resType = hipResourceTypeArray;
    resDescD.res.array.array = cuArrayD;

    hipTextureDesc texDescD;
    memset(&texDescD, 0, sizeof(texDescD));
    texDescD.readMode = hipReadModeElementType;
    texDescD.addressMode[0] = hipAddressModeClamp;
    texDescD.addressMode[1] = hipAddressModeClamp;
    texDescD.filterMode = hipFilterModePoint;
    texDescD.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t texD = 0;
    hipCreateTextureObject(&texD, &resDescD, &texDescD, NULL);

    // Allocate a hipArray in device memory and copy our polylineData there
    hipChannelFormatDesc channelDescP = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipArray* cuArrayP;
    hipMallocArray(&cuArrayP, &channelDescP, imageWidth, imageHeight);
    hipMemcpyToArray(cuArrayP, 0, 0, polylineData, sizePolyline, hipMemcpyHostToDevice);

    hipResourceDesc resDescP;
    memset(&resDescP, 0, sizeof(resDescP));
    resDescP.resType = hipResourceTypeArray;
    resDescP.res.array.array = cuArrayP;

    hipTextureDesc texDescP;
    memset(&texDescP, 0, sizeof(texDescP));
    texDescP.readMode = hipReadModeElementType;
    texDescP.addressMode[0] = hipAddressModeClamp;
    texDescP.addressMode[1] = hipAddressModeClamp;
    texDescP.filterMode = hipFilterModePoint;
    texDescP.normalizedCoords = 1;

    // Create bindless texture object
    hipTextureObject_t texP = 0;
    hipCreateTextureObject(&texP, &resDescP, &texDescP, NULL);

    // Create result array in the device
    T* outputDev;
    hipMalloc(&outputDev, sizeDicom);

    // Create result array in the host
    T* outputHost = (T*)malloc(sizeDicom);

    // Run kernel
    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    rasterizerTest<<<grid, block>>>(outputDev, texD, texP, imageWidth, imageHeight);

    // Copy results to host
    hipMemcpy(outputHost, outputDev, sizeDicom, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroyTextureObject(texD);
    hipDestroyTextureObject(texP);
    hipFreeArray(cuArrayD);
    hipFreeArray(cuArrayP);
    hipFree(outputDev);

    return outputHost;
}

template<typename T, hipChannelFormatKind FK>
__host__ T* modifySurfaceRasterized(int imageWidth, int imageHeight, T* h_dataDicom, unsigned char* h_dataPolyline)
{
    size_t sizeDicom = imageWidth * imageHeight * sizeof(T);
    size_t sizePolyline = imageWidth * imageHeight * sizeof(unsigned char);

    // Create a Surface with our image data and copy that data to the device
    hipChannelFormatDesc channelFormatDicom = hipCreateChannelDesc(8 * sizeof(T), 0, 0, 0, FK);
    hipArray* d_arrayDicom;
    hipMallocArray(&d_arrayDicom, &channelFormatDicom, imageWidth, imageHeight);
    hipMemcpyToArray(d_arrayDicom, 0, 0, h_dataDicom, sizeDicom, hipMemcpyHostToDevice);

    hipResourceDesc resDescDicom;
    memset(&resDescDicom, 0, sizeof(resDescDicom));
    resDescDicom.resType = hipResourceTypeArray;
    resDescDicom.res.array.array = d_arrayDicom;

    hipSurfaceObject_t d_surfDicom = 0;
    hipCreateSurfaceObject(&d_surfDicom, &resDescDicom);


    // Create a surface with our polyline data and copy that data to the device
    hipChannelFormatDesc channelFormatPolyline = hipCreateChannelDesc(8 * 1, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipArray* d_arrayPolyline;
    hipMallocArray(&d_arrayPolyline, &channelFormatPolyline, imageWidth, imageHeight);
    hipMemcpyToArray(d_arrayPolyline, 0, 0, h_dataPolyline, sizePolyline, hipMemcpyHostToDevice);

    hipResourceDesc resDescPolyline;
    memset(&resDescPolyline, 0, sizeof(resDescPolyline));
    resDescPolyline.resType = hipResourceTypeArray;
    resDescPolyline.res.array.array = d_arrayPolyline;

    hipSurfaceObject_t d_surfPolyline = 0;
    hipCreateSurfaceObject(&d_surfPolyline, &resDescPolyline);


    // Create an output surface
    hipArray* d_arrayResult;
    hipMallocArray(&d_arrayResult, &channelFormatDicom, imageWidth, imageHeight);

    hipResourceDesc resDescResult;
    memset(&resDescResult, 0, sizeof(resDescResult));
    resDescResult.resType = hipResourceTypeArray;
    resDescResult.res.array.array = d_arrayResult;

    hipSurfaceObject_t d_surfResult = 0;
    hipCreateSurfaceObject(&d_surfResult, &resDescResult);


    // Run kernel
    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);
    rasterizerTestSurface<<<grid, block>>>(d_surfDicom, d_surfResult, d_surfPolyline, imageWidth, imageHeight);

    // Copy results to host
    T* outputHost = (T*)malloc(sizeDicom);
    hipMemcpyFromArray(outputHost, d_arrayResult, 0, 0, sizeDicom, hipMemcpyDeviceToHost);

    // Cleanup
    hipDestroySurfaceObject(d_surfDicom);
    hipDestroySurfaceObject(d_surfPolyline);
    hipDestroySurfaceObject(d_surfResult);
    hipFreeArray(d_arrayDicom);
    hipFreeArray(d_arrayPolyline);
    hipFreeArray(d_arrayResult);

    return outputHost;
}

//====================================================================================
//TEMPLATE EXPLICIT INSTANTIATIONS
//====================================================================================
//Explicit instantiation so these signatures are available when the linker is linking our lib to the exe
//The compiler would have no way of knowing these specific signatures will be needed since the calls are in
//another compilation unit
template __host__ short* modifyTexture<short, hipChannelFormatKindSigned>(int imageWidth, int imageHeight, short* textureData);
template __host__ unsigned short* modifyTexture<unsigned short, hipChannelFormatKindUnsigned>(int imageWidth, int imageHeight, unsigned short* textureData);
template __host__ short* modifyTextureRasterized<short, hipChannelFormatKindSigned>(int imageWidth, int imageHeight, short* dicomData, unsigned char* polylineData);
template __host__ unsigned short* modifyTextureRasterized<unsigned short, hipChannelFormatKindUnsigned>(int imageWidth, int imageHeight, unsigned short* dicomData, unsigned char* polylineData);
template __host__ short* modifySurfaceRasterized<short, hipChannelFormatKindSigned>(int imageWidth, int imageHeight, short* dicomData, unsigned char* polylineData);
template __host__ unsigned short* modifySurfaceRasterized<unsigned short, hipChannelFormatKindUnsigned>(int imageWidth, int imageHeight, unsigned short* dicomData, unsigned char* polylineData);
