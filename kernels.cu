#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <memory>

#include <kernels.cuh>
#include <gaussian.cuh>
#include <gradient.cuh>
#include <common.cuh>

using namespace std;

//====================================================================================
//KERNELS
//====================================================================================

__global__ void edgeIndicatorKernel(hipSurfaceObject_t gaussInput, hipSurfaceObject_t output)
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float sumX = 0;
    float sumY = 0;
    int index = 0;
    float sample;

    for(int i = -2; i <= 2; i++)
    {
        for(int j = -2; j <= 2; j++)
        {
            surf2Dread(&sample, gaussInput, (x+i)*sizeof(sample), y+j, hipBoundaryModeClamp);

            index = 5*(i+2) + (j+2);
            sumX += sample * d_sobelX[index];
            sumY += sample * d_sobelY[index];
        }
    }

    surf2Dwrite(1.0f / (1.0f + sumX * sumX + sumY * sumY),
                output, x * sizeof(float),
                y,
                hipBoundaryModeClamp);
}

//====================================================================================
//HOST CUDA FUNCTIONS
//====================================================================================

//Forward declarations
__host__ void edgeIndicator(int imageWidth, int imageHeight, float* h_dataDicom, CUDASurface* out_edgeSurf, CUDASurface* out_edgeGradSurf);


__host__ float* applyEdgeIndicator(int imageWidth, int imageHeight, float* h_dicomData)
{
    size_t sizeDicom = imageWidth * imageHeight * sizeof(float);

    hipChannelFormatDesc channelFormatEdge = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipChannelFormatDesc channelFormatEdgeGrad = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
    auto edgeSurf = CUDASurface(nullptr, imageWidth, imageHeight, channelFormatEdge);
    auto edgeGradSurf = CUDASurface(nullptr, imageWidth, imageHeight, channelFormatEdgeGrad);
    edgeSurf.name = "edgeSurf";
    edgeGradSurf.name = "edgeGradSurf";

    edgeIndicator(imageWidth, imageHeight, h_dicomData, &edgeSurf, &edgeGradSurf);

    // Copy results to host memory
    float* h_output = (float*)malloc(sizeDicom);
    eee(hipMemcpyFromArray(h_output, edgeSurf.arr, 0, 0, sizeDicom, hipMemcpyDeviceToHost));

    return h_output;
}

__host__ void edgeIndicator(int imageWidth, int imageHeight, float* h_dataDicom, CUDASurface *out_edgeSurf, CUDASurface *out_edgeGradSurf)
{
    hipChannelFormatDesc channelFormatDicom = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipChannelFormatDesc channelFormatGauss = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    auto dicomSurf = CUDASurface(h_dataDicom, imageWidth, imageHeight, channelFormatDicom);
    auto gaussSurf = CUDASurface(nullptr, imageWidth, imageHeight, channelFormatGauss);
    dicomSurf.name = "dicomSurf";
    gaussSurf.name = "gaussSurf";

    dim3 block(imageWidth / 16, imageHeight / 16,1);
    dim3 grid(16,16,1);

    // Run gaussian kernel
    gaussianKernel<<<grid, block>>>(dicomSurf.surface, gaussSurf.surface);

    // Run edge indicator kernel
    edgeIndicatorKernel<<<grid, block>>>(gaussSurf.surface, out_edgeSurf->surface);

    // Also get the gradient of the edge indicator result
    sobelKernel<<<grid, block>>>(out_edgeSurf->surface, out_edgeGradSurf->surface);


    // The synchronize call will force the host to wait for the kernel to finish. If we don't
    // do this, we might get errors on future checks, but that indicate errors in the kernel, which
    // can be confusing
    eee(hipPeekAtLastError());
    eee(hipDeviceSynchronize());
}

__host__ void initLevelSetData(int imageWidth, int imageHeight, float* h_dicomData, float* h_polylineData, LevelSetData* out_levelSetData)
{
    hipChannelFormatDesc channelFormatPhi = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    out_levelSetData->phi = std::make_unique<CUDASurface>(h_polylineData, imageWidth, imageHeight, channelFormatPhi);
    out_levelSetData->phi->name = "phiSurf";

    hipChannelFormatDesc channelFormatEdge = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    out_levelSetData->edge = std::make_unique<CUDASurface>(nullptr, imageWidth, imageHeight, channelFormatEdge);
    out_levelSetData->edge->name = "edgeSurf";

    hipChannelFormatDesc channelFormatEdgeGrad = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
    out_levelSetData->edgeGrad = std::make_unique<CUDASurface>(nullptr, imageWidth, imageHeight, channelFormatEdgeGrad);
    out_levelSetData->edgeGrad->name = "edgeGradSurf";

    edgeIndicator(imageWidth, imageHeight, h_dicomData, out_levelSetData->edge.get(), out_levelSetData->edgeGrad.get());

    // The synchronize call will force the host to wait for the kernel to finish. If we don't
    // do this, we might get errors on future checks, but that indicate errors in the kernel, which
    // can be confusing
    eee(hipPeekAtLastError());
    eee(hipDeviceSynchronize());
}
